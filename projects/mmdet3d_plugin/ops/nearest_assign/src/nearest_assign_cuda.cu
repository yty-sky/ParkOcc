// Copyright (c) Phigent Robotics. All rights reserved.
// Reference https://arxiv.org/abs/2211.17111


#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

__global__ void nearest_assign_kernel(
                                  const int* l2s_key,
                                  int l2s_size,
                                  const int* occind2detind,
                                  const int *__restrict__ occ_pred,
                                  const int *__restrict__ inst_xyz,
                                  const int *__restrict__ inst_cls,
                                  const int *__restrict__ inst_id_list,
                                  int inst_size,
                                  int* __restrict__ inst_pred) {
  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  // while (idx < 200*200*16)
  if (true)
  {
    int occ_pred_label = occ_pred[idx];
    int dist_min = 100000000;
    for (int index = 0; index < l2s_size; index ++)
    {
      if (occ_pred_label == l2s_key[index])
      {
        int x = idx/(200*16);
        int y = (idx - x*200*16)/16;
        int z = idx - x*200*16 - y*16;
        int inst_ind = 0;
        for (inst_ind = 0; inst_ind < inst_size; inst_ind ++)
        {
          if (inst_cls[inst_ind] == occind2detind[occ_pred_label])
          {
            int dx = x - inst_xyz[inst_ind*3+0];
            int dy = y - inst_xyz[inst_ind*3+1];
            int dz = z - inst_xyz[inst_ind*3+2];
            int dist = dx*dx + dy*dy + dz*dz;
            if (dist < dist_min){
              dist_min = dist;
              inst_pred[idx] = inst_id_list[inst_ind];
            }
          }
        }
        return;
      }
    }
    inst_pred[idx] = occ_pred[idx];

    // idx += blockDim.x * gridDim.x;
  }

}

void nearest_assign(
              const int* l2s_key,
              int l2s_size,
              const int *__restrict__ occind2detind,
              int inst_size,
              const int *__restrict__ occ_pred,
              const int *__restrict__ inst_xyz,
              const int *__restrict__ inst_cls,
              const int *__restrict__ inst_id_list,
              int* __restrict__ inst_pred) {
  // nearest_assign_kernel<<<128, 256>>>(
  nearest_assign_kernel<<<(int)ceil(((double)200 * 200 * 16 / 256)), 256>>>(
    l2s_key, l2s_size, occind2detind, 
    occ_pred, inst_xyz, inst_cls, 
    inst_id_list, inst_size, inst_pred
  );
}


